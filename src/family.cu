#include "hip/hip_runtime.h"
/*
 * family.cu
 * GPU Version
 *
 *  Created on: Mar 7, 2012
 *  Author: Gang Peng <gpeng1@mdanderson.org>
 *
 *  FamSeq is free software. You can redistribute and/or modify it under GNU General Public License
 *  of version 3(GPLv3).
 */

#include <cmath>
#include <algorithm>
#include <numeric>
#include <stdlib.h>
#include <time.h>

#include "family.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

using namespace std;

#define NUM_THREAD 512
#define NUM_BLOCK 8

individual::individual(int id, int gender, int age, int state, int mId, int fId, string sName)
{
	m_id=id;
	m_gender=gender;
	m_age=age;
	m_state=state;
	m_mId=mId;
	m_fId=fId;
	m_sName=sName;
}

bool individual::set_id(int id)
{
	m_id=id;
	return true;
}

bool individual::set_age(int age)
{
	m_age=age;
	return true;
}

bool individual::set_gender(int gender)
{
	m_gender=gender;
	return true;
}

bool individual::set_sate(int state)
{
	m_state=state;
	return true;
}

bool individual::set_mId(int mId)
{
	m_mId=mId;
	return true;
}

bool individual::set_fId(int fId)
{
	m_fId=fId;
	return true;
}

bool individual::set_sName(string sName)
{
	m_sName=sName;
	return true;
}

family::family()
{
}

family::family(const vector<individual> & mem, double mutationRate)
{
	member=mem;
	numInd=member.size();

	//fulfill the whole family
	//realNumInd=numInd;

	//fulfillFamily();

	mRate=mutationRate;

	//set Pr(genotype)
	genoProbN.clear();
	/*
	genoProbN.push_back(0.999*0.999);
	genoProbN.push_back(2*0.999*0.001);
	genoProbN.push_back(0.000001);
	*/
	genoProbN.push_back(0.9985);
	genoProbN.push_back(0.001);
	genoProbN.push_back(0.0005);

	genoProbK.clear();
	genoProbK.push_back(0.45);
	genoProbK.push_back(0.1);
	genoProbK.push_back(0.45);

	genoProbXN.clear();
	genoProbXN.push_back(0.999);
	genoProbXN.push_back(0);
	genoProbXN.push_back(0.001);

	genoProbXK.clear();
	genoProbXK.push_back(0.5);
	genoProbXK.push_back(0);
	genoProbXK.push_back(0.5);

	postProb=dMatrix<double> (numInd,3);
	postProbSingle=dMatrix<double> (numInd,3);

	likelihood=dMatrix<double> (numInd,3);

	//set indicators
	flagInit=false;
	flagLK=false;
	flagPB=false;
	flagPBS=false;
	flagGender=false;
}

family & family::operator =(const family & fm)
{
	if(this==&fm)
	{
		return *this;
	}

	member=fm.get_member();
	numInd=fm.get_numInd();
	realNumInd=fm.get_realNumInd();
	child=fm.get_child();
	parent=fm.get_parent();
	spouse=fm.get_spouse();
	mRate=fm.get_mRate();
	flagInit=fm.get_flagInit();
	flagLK=fm.get_flagLK();
	flagPB=fm.get_flagPB();
	flagPBS=fm.get_flagPBS();
	flagGender=fm.get_flagGender();
	genoProbN=fm.get_genoProbN();
	genoProbK=fm.get_genoProbK();
	genoProbXN=fm.get_genoProbXN();
	genoProbXK=fm.get_genoProbXK();
	pcp2=fm.get_pcp2();
	pcp2Xf=fm.get_pcp2Xf();
	pcp2Xm=fm.get_pcp2Xm();
	postProb=fm.get_postProb(false);
	postProbSingle=fm.get_postProbSingle(false);
	likelihood=fm.get_LK();
	mapP2V=fm.get_mapP2V();
	mapV2P=fm.get_mapV2P();

	return *this;
}

bool family::fulfillFamily()
{
	for(unsigned int i=0;i<realNumInd;i++)
	{
		int fid=member[i].get_fId();
		int mid=member[i].get_mId();

		int indM=-1;
		int indF=-1;
		for(unsigned int j=0;j<realNumInd;j++)
		{
			if(member[j].get_id()==mid)
			{
				indM=j;
			}
			if(member[j].get_id()==fid)
			{
				indF=j;
			}
		}

		if(indM>=0 && indF<0)
		{
			numInd=numInd+1;
			member[i].set_fId(-(int)numInd);
			individual indTmp(-(int)numInd,1,0,0,0,0,"NA");
			member.push_back(indTmp);
		}

		if(indF>=0 && indM<0)
		{
			numInd=numInd+1;
			member[i].set_mId(-((int)numInd));
			individual indTmp(-((int)numInd),2,0,0,0,0,"NA");
			member.push_back(indTmp);
		}
	}
	return true;
}

bool family::checkPed(){
	for(size_t i=0;i<parent.size();i++){
		if(parent[i].size()==2){
			if(member[parent[i][0]].get_gender() != 2){
				cerr<<"Sample "<<member[parent[i][0]].get_id()<<"'s a mother while she is not a female."<<endl;
				return false;
			}

			if(member[parent[i][1]].get_gender() != 1){
				cerr<<"Sample "<<member[parent[i][0]].get_id()<<"'s a father while she is not a male."<<endl;
				return false;
			}
		}
	}
	return true;
}

bool family::init()
{
	if(!setPCP())
	{
		return false;
	}

	if(!setRelation())
	{
		return false;
	}

	if(!checkPed()){
		return false;
	}

	return true;
}

bool family::set_mRate(double rate)
{
	if(mRate!=rate)
	{
		mRate=rate;
		if(!setPCP())
		{
			return false;
		}
	}
	return true;
}

bool family::set_lc(double lc)
{
	m_lc = lc;
	return true;
}

bool family::setPCP()
{
	//pcp2
	if(!calPCP2())
	{
		return false;
	}

	if(!calPCP2Xf())
	{
		return false;
	}

	if(!calPCP2Xm())
	{
		return false;
	}

	/*
	for(size_t i=0;i<pcp2Xf.size();i++)
	{
		cout<<pcp2Xf[i]<<endl;
	}

	for(size_t i=0;i<pcp2Xm.size();i++)
	{
		cout<<pcp2Xm[i]<<endl;
	}
	*/
	return true;
}

bool family::setRelation()
{
	child.clear();
	parent.clear();
	spouse.clear();

	child.resize(numInd);
	parent.resize(numInd);
	spouse.resize(numInd);

	for(unsigned int i=0;i<numInd;i++)
	{
		int mId=member[i].get_mId();
		int fId=member[i].get_fId();
		int indM=-1;
		int indF=-1;
		for(unsigned int j=0;j<numInd;j++)
		{
			if(mId==member[j].get_id())
			{
				indM=j;
			}
			if(fId==member[j].get_id())
			{
				indF=j;
			}
		}

		if((indM<0 && indF>=0) || (indM>=0 && indF<0))
		{
			cout<<"This is not a fulfill family. Please check the ped file."<<endl;
			return false;
		}

		if(indM>=0 && indF>=0)
		{
			//first one is mother
			//second one is father
			parent[i].push_back(indM);
			parent[i].push_back(indF);
			child[indM].push_back(i);
			child[indF].push_back(i);
			bool notFind=true;
			for(vector<int>::size_type j=0;j<spouse[indM].size();j++)
			{
				if(indF==spouse[indM][j])
				{
					notFind=false;
					break;
				}
			}
			if(notFind)
			{
				spouse[indM].push_back(indF);
				spouse[indF].push_back(indM);
			}
		}
	}
	return true;
}

bool family::set_fg(bool fg)
{
	flagGender=fg;
	return true;
}

bool family::set_mapP2V(vector<int> mP2V)
{
	mapP2V=mP2V;
	return true;
}

bool family::set_mapV2P(vector<int> mV2P)
{
	mapV2P=mV2P;
	realNumInd=0;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			realNumInd++;
		}
	}
	return true;
}

bool family::calPCP2()
{
	return calPCP2S(2,pcp2);
}

bool family::calPCP2Xf()
{
	pcp2Xf.clear();
	for(int i=0;i<3;i++)
	{
		dMatrix<double> dMTmp(3,3,0);
		pcp2Xf.push_back(dMTmp);
	}

	pcp2Xf[0](0,0)=(1.0-mRate)*(1.0-mRate);
	pcp2Xf[1](0,0)=2*mRate*(1.0-mRate);
	pcp2Xf[2](0,0)=mRate*mRate;

	pcp2Xf[0](0,2)=(1.0-mRate)*mRate;
	pcp2Xf[1](0,2)=(1.0-mRate)*(1.0-mRate)+mRate*mRate;
	pcp2Xf[2](0,2)=(1.0-mRate)*mRate;

	pcp2Xf[0](1,0)=(1.0-mRate)*(1.0-mRate)/2+mRate*(1.0-mRate)/2;
	pcp2Xf[1](1,0)=mRate*(1.0-mRate)+(1.0-mRate)*(1.0-mRate)/2+mRate*mRate/2;
	pcp2Xf[2](1,0)=mRate*mRate/2+mRate*(1.0-mRate)/2;

	pcp2Xf[0](1,2)=mRate*mRate/2+mRate*(1.0-mRate)/2;
	pcp2Xf[1](1,2)=mRate*(1.0-mRate)+(1.0-mRate)*(1.0-mRate)/2+mRate*mRate/2;
	pcp2Xf[2](1,2)=(1.0-mRate)*(1.0-mRate)/2+mRate*(1.0-mRate)/2;

	pcp2Xf[0](2,0)=(1.0-mRate)*mRate;
	pcp2Xf[1](2,0)=(1.0-mRate)*(1.0-mRate)+mRate*mRate;
	pcp2Xf[2](2,0)=(1.0-mRate)*mRate;

	pcp2Xf[0](2,2)=mRate*mRate;
	pcp2Xf[1](2,2)=2*mRate*(1.0-mRate);
	pcp2Xf[2](2,2)=(1.0-mRate)*(1.0-mRate);
	return true;
}

bool family::calPCP2Xm()
{
	pcp2Xm.clear();
	for(int i=0;i<3;i++)
	{
		dMatrix<double> dMTmp(3,3,0);
		pcp2Xm.push_back(dMTmp);
	}

	pcp2Xm[0](0,0)=1-mRate;
	pcp2Xm[2](0,0)=mRate;

	pcp2Xm[0](0,2)=1-mRate;
	pcp2Xm[2](0,2)=mRate;

	pcp2Xm[0](1,0)=0.5;
	pcp2Xm[2](1,0)=0.5;

	pcp2Xm[0](1,2)=0.5;
	pcp2Xm[2](1,2)=0.5;

	pcp2Xm[0](2,0)=mRate;
	pcp2Xm[2](2,0)=1-mRate;

	pcp2Xm[0](2,2)=mRate;
	pcp2Xm[2](2,2)=1-mRate;
	return true;
}

bool family::calPCP2S(int nAllele, std::vector<dMatrix<double> > & PCP2S)
{
	int nGeno=(nAllele+1)*nAllele/2;
	PCP2S.clear();
	for(int i=0;i<nGeno;i++)
	{
		dMatrix<double> dMTmp(nGeno,nGeno,0);
		PCP2S.push_back(dMTmp);
	}

	dMatrix<int> codeTable(nAllele,nAllele,0);
	dMatrix<int> decodeTable(nGeno,2,0);
	int count=0;
	for(int i=0;i<nAllele;i++)
	{
		for(int j=i;j<nAllele;j++)
		{
			codeTable(i,j)=count;
			codeTable(j,i)=count;
			decodeTable(count,0)=i;
			decodeTable(count,1)=j;
			count++;
		}
	}

	if(mRate==0)
	{
		for(int i=0;i<nGeno;i++)
		{
			for(int j=0;j<nGeno;j++)
			{
				//haplotype
				int i0,i1,j0,j1;
				i0=decodeTable(i,0);
				i1=decodeTable(i,1);
				j0=decodeTable(j,0);
				j1=decodeTable(j,1);
				PCP2S[codeTable(i0,j0)](i,j)=PCP2S[codeTable(i0,j0)](i,j)+0.25;
				PCP2S[codeTable(i0,j1)](i,j)=PCP2S[codeTable(i0,j1)](i,j)+0.25;
				PCP2S[codeTable(i1,j0)](i,j)=PCP2S[codeTable(i1,j0)](i,j)+0.25;
				PCP2S[codeTable(i1,j1)](i,j)=PCP2S[codeTable(i1,j1)](i,j)+0.25;
			}
		}
	}
	else
	{
		for(int i=0;i<nGeno;i++)
		{
			for(int j=0;j<nGeno;j++)
			{
				vector<double> p1(nAllele,mRate/(2*(nAllele-1)));
				vector<double> p2(nAllele,mRate/(2*(nAllele-1)));

				//haplotype 0 and 0
				p1[decodeTable(i,0)]=(1-mRate)/2;
				p2[decodeTable(j,0)]=(1-mRate)/2;
				for(int k=0;k<nAllele;k++)
				{
					for(int l=0;l<nAllele;l++)
					{
						PCP2S[codeTable(k,l)](i,j)=PCP2S[codeTable(k,l)](i,j)+p1[k]*p2[l];
					}
				}

				//haplotype 0 and 1
				p2[decodeTable(j,0)]=mRate/(2*(nAllele-1));
				p2[decodeTable(j,1)]=(1-mRate)/2;
				for(int k=0;k<nAllele;k++)
				{
					for(int l=0;l<nAllele;l++)
					{
						PCP2S[codeTable(k,l)](i,j)=PCP2S[codeTable(k,l)](i,j)+p1[k]*p2[l];
					}
				}

				//haplotype 1 and 0
				p1[decodeTable(i,0)]=mRate/(2*(nAllele-1));
				p2[decodeTable(j,1)]=mRate/(2*(nAllele-1));
				p1[decodeTable(i,1)]=(1-mRate)/2;
				p2[decodeTable(j,0)]=(1-mRate)/2;
				for(int k=0;k<nAllele;k++)
				{
					for(int l=0;l<nAllele;l++)
					{
						PCP2S[codeTable(k,l)](i,j)=PCP2S[codeTable(k,l)](i,j)+p1[k]*p2[l];
					}
				}

				//haplotype 1 and 1
				p2[decodeTable(j,0)]=mRate/(2*(nAllele-1));
				p2[decodeTable(j,1)]=(1-mRate)/2;
				for(int k=0;k<nAllele;k++)
				{
					for(int l=0;l<nAllele;l++)
					{
						PCP2S[codeTable(k,l)](i,j)=PCP2S[codeTable(k,l)](i,j)+p1[k]*p2[l];
					}
				}
			}
		}
	}

	return true;
}

bool family::set_genoProbK(const vector<double> & prob)
{
	genoProbK=prob;
	return true;
}

bool family::set_genoProbN(const vector<double> & prob)
{
	genoProbN=prob;
	return true;
}

bool family::set_genoProbXK(const vector<double> & prob)
{
	genoProbXK=prob;
	return true;
}

bool family::set_genoProbXN(const vector<double> & prob)
{
	genoProbXN=prob;
	return true;
}

dMatrix<double> family::get_postProb(bool flag) const
{
	if(flag)
	{
		if(!flagPB)
		{
			cout<<"The posterior probability (FamSeqPro) hasn't been calculated yet!"<<endl;
			return dMatrix<double> (1,1,-1);
		}
		dMatrix<double> rlt(realNumInd,3);
		int ind=0;
		for(size_t i=0;i<mapV2P.size();i++)
		{
			if(mapV2P[i]>=0)
			{
				for(int j=0;j<3;j++)
				{
					rlt(ind,j)=postProb(mapV2P[i],j);
				}
				ind++;
			}
		}
		return rlt;
	}
	else
	{
		return postProb;
	}
}

dMatrix<double> family::get_postProbSingle(bool flag) const
{
	if(flag)
	{
		if(!flagPBS)
		{
			cout<<"The posterior probability (Single) hasn't been calculated yet!"<<endl;
			return dMatrix<double> (1,1,-1);
		}
		dMatrix<double> rlt(realNumInd,3);
		int ind=0;
		for(size_t i=0;i<mapV2P.size();i++)
		{
			if(mapV2P[i]>=0)
			{
				for(int j=0;j<3;j++)
				{
					rlt(ind,j)=postProbSingle(mapV2P[i],j);
				}
				ind++;
			}
		}
		return rlt;
	}
	else
	{
		return postProbSingle;
	}
}

vector<int> family::get_postRlt() const
{
	if(!flagPB)
	{
		cout<<"The posterior probability (FamSeqPro) hasn't been calculated yet!"<<endl;
		vector<int> tmp;
		return tmp;
	}

	vector<int> rlt;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double bigTmp=-1;
			int ind=-1;
			for(int j=0;j<3;j++)
			{
				if(bigTmp<postProb(mapV2P[i],j))
				{
					bigTmp=postProb(mapV2P[i],j);
					ind=j;
				}
			}
			rlt.push_back(ind);
		}
	}

	return rlt;
}

vector<int> family::get_postRltSingle() const
{
	if(!flagPBS)
	{
		cout<<"The posterior probability (Single) hasn't been calculated yet!"<<endl;
		vector<int> tmp;
		return tmp;
	}

	vector<int> rlt;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double bigTmp=-1;
			int ind=-1;
			for(int j=0;j<3;j++)
			{
				if(bigTmp<postProbSingle(mapV2P[i],j))
				{
					bigTmp=postProbSingle(mapV2P[i],j);
					ind=j;
				}
			}
			rlt.push_back(ind);
		}
	}

	return rlt;
}

bool family::set_LK(const dMatrix<double> & lk)
{
	/*
	if(lk.get_row()!=int(numInd))
	{
		for(int i=0;i<lk.get_row();i++)
		{
			for(int j=0;j<3;j++)
			{
				likelihood(i,j)=lk(i,j);
			}
		}
		for(unsigned int i=lk.get_row();i<numInd;i++)
		{
			for(int j=0;j<3;j++)
			{
				likelihood(i,j)=1;
			}
		}
	}
	else
	{
		for(int i=0;i<lk.get_row();i++)
		{
			for(int j=0;j<3;j++)
			{
				likelihood(i,j)=lk(i,j);
			}
		}
	}
	*/

	if(int(numInd)!=lk.get_row() || lk.get_column()!=3)
	{
		cout<<"The dimention of likelihood matrix is wrong. Cannot set likelihood."<<endl;
		return false;
	}

	for(int i=0;i<lk.get_row();i++)
	{
		for(int j=0;j<3;j++)
		{
			likelihood(i,j)=lk(i,j);
		}
	}

	flagLK=true;
	flagPB=false;
	flagPBS=false;
	return true;
}

bool decodeGeno(int genoCode, int numSample, vector<int> & geno){
	geno.resize(numSample);
	for(int i=0;i<numSample;i++){
		int rm = genoCode%3;
		geno[i] = rm;
		genoCode = (genoCode-rm)/3;
	}
	return true;
}


extern "C" __global__ void calPostProb(int* father, int* mother, double* lk, double* pcp2, double* postProb, double * genoFry, int numSample,int numGeno){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	double pTmp[60]={0};
	for(int i=bid*NUM_THREAD+tid;i<numGeno;i+=NUM_BLOCK*NUM_THREAD){
		int geno[20];
		int genoCode=i;
		for(int j=0;j<numSample;j++){
			int rm=genoCode%3;
			geno[j]=rm;
			genoCode=(genoCode-rm)/3;
		}

		double postTmp = 10000000;
		for(int j=0;j<numSample;j++){
			if(father[j]<0){
				postTmp = postTmp*genoFry[geno[j]]*lk[j*3+geno[j]];
			}
			else{
				int indexpcp = geno[j]*9+geno[mother[j]]*3+geno[father[j]];
				postTmp = postTmp*pcp2[indexpcp]*lk[j*3+geno[j]];
			}
		}
		for(int j=0;j<numSample;j++){
			pTmp[j*3+geno[j]] += postTmp;
		}
	}

	for(int i=0;i<numSample*3;i++){
		postProb[(bid*NUM_THREAD+tid)*numSample*3+i]=pTmp[i];
	}
}



/*
extern "C" __global__ void calPostProb(int* father, int* mother, double* lk, double* pcp2, double* postProb, double * genoFry, int numSample,int numGeno){
	for(int i=blockIdx.x;i<numGeno;i+=NUM_BLOCK){
		int geno[20];
		int genoCode=i;
		for(int j=0;j<numSample;j++){
			int rm=genoCode%3;
			geno[j]=rm;
			genoCode=(genoCode-rm)/3;
		}

		double postTmp = 10000000;
		for(int j=0;j<numSample;j++){
			if(father[j]<0){
				postTmp = postTmp*genoFry[geno[j]]*lk[j*3+geno[j]];
			}
			else{
				int indexpcp = geno[j]*9+geno[mother[j]]*3+geno[father[j]];
				postTmp = postTmp*pcp2[indexpcp]*lk[j*3+geno[j]];
			}
		}

		for(int j=0;j<numSample;j++){
			postProb[blockIdx.x*numSample*3+j*3+geno[j]]=postTmp+postProb[blockIdx.x*numSample*3+j*3+geno[j]];
		}
	}
}
*/


/*
extern "C" __global__ void calPostProb(int* father, int* mother, double * lk, double * pcp2, double * postProb, double * genoFry, int numSample, int numGeno){
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < numGeno){
		int geno[20];
		int genoCode = index;
		for(int i=0;i<numSample;i++){
			int rm = genoCode%3;
			geno[i] = rm;
			genoCode = (genoCode-rm)/3;
		}
		double postTmp = 10000000;
		for(int i=0;i<numSample;i++){
			if(father[i]<0){
				postTmp = postTmp*genoFry[geno[i]]*lk[i*3+geno[i]];
			}
			else{
				int indexpcp = geno[i]*9+geno[mother[i]]*3+geno[father[i]];
				postTmp = postTmp*pcp2[indexpcp]*lk[i*3+geno[i]];
			}
		}

		//for(int i=0;i<numSample;i++){
			//postProb[i*3+geno[i]]=postProb[i*3+geno[i]]+1;
			//atomicAdd((postProb+(i*3+geno[i])),float(postTmp));
			//atomicAdd((int*) postProb,int(postTmp));

		//}

		postProb[index] = postTmp;
		//postProb[index] = postTmp;
	}
}
*/

/*
extern "C" __global__ void sumPostProb(int numGeno, int numSample, double * postProb, double * postProbS){
	for(int i=0;i<numGeno;i++){
		int geno[100];
		int genoCode = i;
		for(int j=0;j<numSample;j++){
			int rm = genoCode%3;
			geno[j] = rm;
			genoCode = (genoCode-rm)/3;
		}

		for(int j=0;j<numSample;j++){
			postProbS[j*3+geno[j]]=postProbS[j*3+geno[j]]+postProb[i];
		}
	}
}
*/


extern "C" __global__ void sumPostProb(int numGeno, int numSample, double * postProb, double * postProbS){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	double pTmp[60];
	for(int i=bid*NUM_THREAD+tid;i<numGeno;i+=NUM_BLOCK*NUM_THREAD){
		int geno[20];
		int genoCode=i;
		for(int j=0;j<numSample;j++){
			int rm=genoCode%3;
			geno[j]=rm;
			genoCode=(genoCode-rm)/3;
		}
		for(int j=0;j<numSample;j++){
			pTmp[j*3+geno[j]]+=postProb[i];
		}
	}
	for(int i=0;i<numSample;i++){
		for(int j=0;j<3;j++){
			postProbS[(bid*NUM_THREAD+tid)*numSample*3+i*3+j] = pTmp[i*3+j];
		}
	}

	/*
	for(int i=blockIdx.x;i<numGeno;i+=NUM_BLOCK){
		int geno[20];
		int genoCode=i;
		for(int j=0;j<numSample;j++){
			int rm=genoCode%3;
			geno[j]=rm;
			genoCode=(genoCode-rm)/3;
		}
		for(int j=0;j<numSample;j++){
			postProbS[blockIdx.x*numSample*3+j*3+geno[j]] = postProbS[blockIdx.x*numSample*3+j*3+geno[j]]+postProb[i];
		}
	}
	*/
}


extern "C" __global__ void calPostProbX(int * gender, int* father, int* mother, double * lk, double * pcp2M, double* pcp2F, double * postProb, double * genoFryM, double* genoFryF, int numSample, int numGeno){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	double pTmp[60]= {0};
	for(int i=bid*NUM_THREAD+tid;i<numGeno;i+=NUM_BLOCK*NUM_THREAD){
		int geno[20];
		int genoCode=i;
		for(int j=0;j<numSample;j++){
			int rm=genoCode%3;
			geno[j]=rm;
			genoCode=(genoCode-rm)/3;
		}

		double postTmp = 10000000;
		for(int j=0;j<numSample;j++){
			if(father[j]<0){
				if(gender[j]==1){
					postTmp = postTmp*genoFryM[geno[j]]*lk[j*3+geno[j]];
				}
				else{
					postTmp = postTmp*genoFryF[geno[j]]*lk[j*3+geno[j]];
				}
			}
			else{
				int indexpcp = geno[j]*9+geno[mother[j]]*3+geno[father[j]];
				if(gender[j]==1){
					postTmp = postTmp*pcp2M[indexpcp]*lk[j*3+geno[j]];
				}
				else{
					postTmp = postTmp*pcp2F[indexpcp]*lk[j*3+geno[j]];
				}	
			}
		}
		
		for(int j=0;j<numSample;j++){
			pTmp[j*3+geno[j]] += postTmp;
		}
	}
		
	for(int i=0;i<numSample*3;i++){
		postProb[(bid*NUM_THREAD+tid)*numSample*3+i]=pTmp[i];
	}
}

#if 1
bool family::calPostProbBN(bool Known, int chrType)
{
	if(!flagLK)
	{
		cout<<"Likelihood has not been set. Please set likelihood first."<<endl;
		return false;
	}

	if(!calPostProbSingle(Known,chrType))
	{
		//cout<<Known<<'\t'<<chrType<<endl;
		//cout<<likelihood<<endl;
		return false;
	}

	//cout<<likelihood<<endl;

	bool flagSingle=true;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double big=0;
			double sum=0;
			for(int j=0;j<3;j++)
			{
				if(big<likelihood(mapV2P[i],j))
				{
					big=likelihood(mapV2P[i],j);
				}
				sum=sum+likelihood(mapV2P[i],j);
			}
			big=big/sum;
			if(big<=m_lc)
			{
				flagSingle=false;
				break;
			}
		}
	}

	postProb.setZero();

	if(flagSingle)
	{
		if(chrType==0)
		{
			vector<double> genoProb;
			if(Known)
			{
				genoProb=genoProbK;
			}
			else
			{
				genoProb=genoProbN;
			}
			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					postProb(i,j)=likelihood(i,j)*genoProb[j];
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		else if(chrType==1)
		{
			vector<double> genoProbM;
			vector<double> genoProbF;
			if(Known)
			{
				genoProbM=genoProbXK;
			}
			else
			{
				genoProbM=genoProbXN;
			}
			if(Known)
			{
				genoProbF=genoProbK;
			}
			else
			{
				genoProbF=genoProbN;
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					if(member[i].get_gender()==1)
					{
						postProb(i,j)=likelihood(i,j)*genoProbM[j];
					}
					else
					{
						postProb(i,j)=likelihood(i,j)*genoProbF[j];
					}
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		flagPB=true;
		return true;
	}



	if(chrType==0)
	{
		/*
		set up data that will be used in BN
		*/
		vector<int> host_father(numInd,-1);
		vector<int> host_mother(numInd,-1);
		for(size_t i=0;i<numInd;i++){
			if(parent[i].size()!=0){
				host_mother[i]=parent[i][0];
				host_father[i]=parent[i][1];
			}
		}

		vector<double> host_lk(numInd*3,0);
		for(size_t i=0;i<numInd;i++){
			host_lk[i*3] = likelihood(i,0);
			host_lk[i*3+1] = likelihood(i,1);
			host_lk[i*3+2] = likelihood(i,2);
		}

		vector<double> host_pcp2(27,0);
		for(int i=0;i<3;i++){
			for(int j=0;j<3;j++){
				for(int k=0;k<3;k++){
					host_pcp2[i*9+j*3+k] = pcp2[i](j,k);
				}
			}
		}

		int numGeno = (int)pow(3.0,(double)numInd);

		hipError_t cudaStatus;

		int * dev_father = 0;
		int * dev_mother = 0;
		double * dev_lk = 0;
		double * dev_pcp2 =0;
		//double * dev_postProb = 0;
		double * dev_genoFry = 0;
		double * dev_postProb = 0;
		//double * dev_postProbS = 0;


		
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		// Allocate GPU buffers
		cudaStatus = hipMalloc((void**)&dev_father, host_father.size() * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_mother, host_mother.size() * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_lk, host_lk.size() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);
			
			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_pcp2, host_pcp2.size() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		/*
		cudaStatus = hipMalloc((void**)&dev_postProb, (int)pow(3.0,(double)numInd) * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			delete host_postProb;
			
			flagPB=false;
			return false;
		}
		*/

		cudaStatus = hipMalloc((void**)&dev_genoFry, 3*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		
		cudaStatus = hipMalloc((void**)&dev_postProb, (NUM_THREAD*NUM_BLOCK)*3*numInd*sizeof(double));
		//cudaStatus = hipMalloc((void**)&dev_postProb, numGeno*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);
			
			flagPB=false;
			return false;
		}
		

		/*
		cudaStatus = hipMalloc((void**)&dev_postProbS, (NUM_THREAD*NUM_BLOCK)*3*numInd*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			delete host_postProb;
			
			flagPB=false;
			return false;
		}
		*/
		
		

		/*
		copy data
		*/
		cudaStatus = hipMemcpy(dev_father, &host_father[0], host_father.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed father!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);
			
			flagPB=false;
			return false;
		}
		
		/*
		int *test_father = new int[host_father.size()];
		hipMemcpy(test_father, dev_father, host_father.size()*sizeof(int), hipMemcpyDeviceToHost);
		for(size_t i =0; i<host_father.size();i++){
			cout<<host_father[i]<<'\t'<<test_father[i]<<endl;
		}
		delete test_father;
		*/
		

		cudaStatus = hipMemcpy(dev_mother, &host_mother[0], host_mother.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed mother!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		/*
		int *test_mother = new int[host_mother.size()];
		hipMemcpy(test_mother, dev_mother, host_mother.size()*sizeof(int), hipMemcpyDeviceToHost);
		for(size_t i =0; i<host_mother.size();i++){
			cout<<host_mother[i]<<'\t'<<test_mother[i]<<endl;
		}
		delete test_mother;
		*/

		cudaStatus = hipMemcpy(dev_lk, &host_lk[0], host_lk.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed lk!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		/*
		double* test_lk = new double[host_lk.size()];
		hipMemcpy(test_lk, dev_lk, host_lk.size()*sizeof(double), hipMemcpyDeviceToHost);
		for(size_t i=0;i<host_lk.size();i++){
			cout<<test_lk[i]<<'\t'<<host_lk[i]<<endl;
		}
		delete test_lk;
		*/
		
		
		
		cudaStatus = hipMemcpy(dev_pcp2, &host_pcp2[0], host_pcp2.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed pcp2!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}
		

		/*
		double* test_pcp2 = new double[host_pcp2.size()];
		hipMemcpy(test_pcp2, dev_pcp2, host_pcp2.size()*sizeof(double),hipMemcpyDeviceToHost);
		for(size_t i=0;i<host_pcp2.size();i++)
		{
			cout<<test_pcp2[i]<<'\t'<<host_pcp2[i]<<endl;
		}
		delete test_pcp2;
		*/

		if(Known){
			cudaStatus = hipMemcpy(dev_genoFry, &genoProbK[0], 3*sizeof(double), hipMemcpyHostToDevice);

			/*
			double* test_genoFre = new double[3];
			hipMemcpy(test_genoFre, dev_genoFry, 3*sizeof(double), hipMemcpyDeviceToHost);
			for(size_t i=0;i<3;i++){
				cout<<test_genoFre[i]<<'\t'<<genoProbK[i]<<endl;
			}
			delete test_genoFre;
			*/
		}
		else{
			cudaStatus = hipMemcpy(dev_genoFry, &genoProbN[0], 3*sizeof(double), hipMemcpyHostToDevice);

			/*
			double* test_genoFre = new double[3];
			hipMemcpy(test_genoFre, dev_genoFry, 3*sizeof(double), hipMemcpyDeviceToHost);
			for(size_t i=0;i<3;i++){
				cout<<test_genoFre[i]<<'\t'<<genoProbN[i]<<endl;
			}
			delete test_genoFre;
			*/
		}
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed PenoProb!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			
			flagPB=false;
			return false;
		}

		
		/*
		for(int i=0;i<3*numInd*NUM_BLOCK;i++){
			host_postProb[i]=0;
		}

		//for(int i=0;i<3*numInd;i++)
		//{
		//	cout<<host_postProb[i]<<endl;
		//}

		cudaStatus = hipMemcpy(dev_postProbS, host_postProb, 3*numInd*NUM_BLOCK*sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed host_postProb!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			delete host_postProb;
			
			flagPB=false;
			return false;
		}
		*/

		//cudaStatus = hipMemset(dev_postProb, 0, (NUM_THREAD*NUM_BLOCK)*3*numInd*sizeof(double));
		

		//for(int i=0;i<3*numInd*NUM_THREAD;i++)
		//{
		//	cout<<host_postProb[i]<<endl;
		//}
		//for(int i=0;i<3*numInd;i++)
		//{
		//	host_posteriorProb[i]=0;
		//}


		//int numThread = 512;
		//calPostProb<<<(numGeno + numThread-1)/numThread, numThread>>>(dev_father, dev_mother, dev_lk, dev_pcp2, dev_postProb, dev_genoFry, numInd, numGeno);

		//calPostProb<<<(numGeno + NUM_THREAD-1)/NUM_THREAD, NUM_THREAD>>>(dev_father, dev_mother, dev_lk, dev_pcp2, dev_postProb, dev_genoFry, numInd, numGeno);

/*
		// Allocate CUDA events that we'll use for timing
		hipError_t error;
		hipEvent_t start;
		error = hipEventCreate(&start);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		hipEvent_t stop;
		error = hipEventCreate(&stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(start, NULL);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		calPostProb<<<NUM_BLOCK, NUM_THREAD>>>(dev_father, dev_mother, dev_lk, dev_pcp2, dev_postProb, dev_genoFry, numInd, numGeno);
		
		// Record the stop event
		error = hipEventRecord(stop, NULL);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Wait for the stop event to complete
		error = hipEventSynchronize(stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		float msecTotal = 0.0f;
		error = hipEventElapsedTime(&msecTotal, start, stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Compute and print the performance
		printf("Time= %.3f msec\n",msecTotal);

*/
		//sumPostProb<<<1,1>>>(dev_postProb, dev_posteriorProb, numGeno, numInd);

		//sumPostProb<<<NUM_BLOCK,NUM_THREAD>>>(numGeno, numInd, dev_postProb, dev_postProbS);

		calPostProb<<<NUM_BLOCK, NUM_THREAD>>>(dev_father, dev_mother, dev_lk, dev_pcp2, dev_postProb, dev_genoFry, numInd, numGeno);
		
		double * host_postProb = new double[NUM_THREAD*NUM_BLOCK*3*numInd];
		cudaStatus = hipMemcpy(host_postProb, dev_postProb, NUM_THREAD*NUM_BLOCK*3*numInd*sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed p to p!");
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2);
			//hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFry);
			hipFree((void*)dev_postProb);
			//hipFree((void*)dev_postProbS);

			delete host_postProb;
			
			flagPB=false;
			return false;
		}
		

		postProb.setZero();

		for(int i=0;i<NUM_BLOCK*NUM_THREAD;i++){
			for(int j=0;j<numInd;j++){
				for(int k=0;k<3;k++){
					postProb(j,k)=postProb(j,k)+host_postProb[i*numInd*3+j*3+k];
				}
			}
		}
		
		/*
		for(int i=0;i<NUM_BLOCK;i++){
			for(int j=0;j<numInd;j++){
				for(int k=0;k<3;k++){
					//postProb(j,k)=postProb(j,k)+host_postProb[i*3*numInd+j*3+k];
					postProb(j,k)=1;
				}
			}
		}
		*/
		
		

		/*
		int geno[20]={0};
		double* pTmp = new double[numInd*3];
		for(int i=0;i<numInd*3;i++){
			pTmp[i]=0;
		}
		for(int i=0;i<numGeno;i++){
			
			for(int j=0;j<numInd;j++){
				geno[j]++;
				if(geno[j]==3){
					geno[j]=0;
				}
				else{
					break;
				}
			}
			
			for(int j=0;j<numInd;j++){
				//postProb(j,geno[j])=postProb(j,geno[j])+host_postProb[i];
				pTmp[j*3+geno[j]] = pTmp[j*3+geno[j]]+host_postProb[i];
			}
		}
		
		for(int i=0;i<numInd;i++){
			for(int j=0;j<3;j++){
				postProb(i,j)=pTmp[i*3+j];
			}
		}
		delete[] pTmp;
		*/
		
		

		//for(int i=0;i<3*numInd*NUM_THREAD;i++)
		//{
		//	cout<<host_postProb[i]<<endl;
		//}
		/*
		for(unsigned int i=0;i<numInd;i++){
			for(int j=0;j<3;j++){
				//postProb(i,j) = host_postProb[i*3+j];
				postProb(i,j) = 1;
			}
		}
		*/

		

		for(unsigned int i=0;i<numInd;i++){
			double sTmp = 0;
			for(int j=0;j<3;j++){
				sTmp = sTmp + postProb(i,j);
				//cout<<postProb(i,j)<<'\t';
			}
			//cout<<endl;
			if(sTmp<=0)
			{
				return false;
			}
			else
			{
				for(int j=0;j<3;j++){
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		
		hipFree((void*)dev_father);
		hipFree((void*)dev_mother);
		hipFree((void*)dev_lk);
		hipFree((void*)dev_pcp2);
		//hipFree((void*)dev_postProb);
		hipFree((void*)dev_genoFry);
		hipFree((void*)dev_postProb);
		//hipFree((void*)dev_postProbS);

		delete host_postProb;

	}
	else if(chrType==1)
	{
		vector<double> genoProbM;
		vector<double> genoProbF;
		if(Known)
		{
			genoProbM=genoProbXK;
		}
		else
		{
			genoProbM=genoProbXN;
		}
		if(Known)
		{
			genoProbF=genoProbK;
		}
		else
		{
			genoProbF=genoProbN;
		}

		/*
		set up data that will be used in BN
		*/
		vector<int> host_gender(numInd);
		for(size_t i=0;i<numInd;i++){
			host_gender[i] = member[i].get_gender();
		}

		vector<int> host_father(numInd,-1);
		vector<int> host_mother(numInd,-1);
		for(size_t i=0;i<numInd;i++){
			if(parent[i].size()!=0){
				host_mother[i]=parent[i][0];
				host_father[i]=parent[i][1];
			}
		}

		vector<double> host_lk(numInd*3,0);
		for(size_t i=0;i<numInd;i++){
			host_lk[i*3] = likelihood(i,0);
			host_lk[i*3+1] = likelihood(i,1);
			host_lk[i*3+2] = likelihood(i,2);
		}

		vector<double> host_pcp2M(27,0);
		for(int i=0;i<3;i++){
			for(int j=0;j<3;j++){
				for(int k=0;k<3;k++){
					host_pcp2M[i*9+j*3+k] = pcp2Xm[i](j,k);
				}
			}
		}

		vector<double> host_pcp2F(27,0);
		for(int i=0;i<3;i++){
			for(int j=0;j<3;j++){
				for(int k=0;k<3;k++){
					host_pcp2F[i*9+j*3+k] = pcp2Xf[i](j,k);
				}
			}
		}

		int numGeno = (int)pow(3.0,(double)numInd);

		hipError_t cudaStatus;

		int * dev_gender = 0;
		int * dev_father = 0;
		int * dev_mother = 0;
		double * dev_lk = 0;
		//male
		double * dev_pcp2M =0;
		//female
		double * dev_pcp2F = 0;
		//double * dev_posteriorProb = 0;
		//male
		double * dev_genoFryM = 0;
		//female
		double * dev_genoFryF = 0;
		double * dev_postProb = 0;

		
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		// Allocate GPU buffers
		cudaStatus = hipMalloc((void**)&dev_gender, host_gender.size() * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_father, host_father.size() * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_mother, host_mother.size() * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_lk, host_lk.size() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_pcp2M, host_pcp2M.size() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);


			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_pcp2F, host_pcp2F.size() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_postProb, (NUM_THREAD*NUM_BLOCK)*3*numInd*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_genoFryF, 3*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMalloc((void**)&dev_genoFryM, 3*sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}


		/*
		copy data
		*/
		cudaStatus = hipMemcpy(dev_gender, &host_gender[0], host_gender.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMemcpy(dev_father, &host_father[0], host_father.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}
		
		/*
		int *test_father = new int[host_father.size()];
		hipMemcpy(test_father, dev_father, host_father.size()*sizeof(int), hipMemcpyDeviceToHost);
		for(size_t i =0; i<host_father.size();i++){
			cout<<host_father[i]<<'\t'<<test_father[i]<<endl;
		}
		delete test_father;
		*/
		
		

		cudaStatus = hipMemcpy(dev_mother, &host_mother[0], host_mother.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		/*
		int *test_mother = new int[host_mother.size()];
		hipMemcpy(test_mother, dev_mother, host_mother.size()*sizeof(int), hipMemcpyDeviceToHost);
		for(size_t i =0; i<host_mother.size();i++){
			cout<<host_mother[i]<<'\t'<<test_mother[i]<<endl;
		}
		delete test_mother;
		*/
		

		cudaStatus = hipMemcpy(dev_lk, &host_lk[0], host_lk.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		/*
		double* test_lk = new double[host_lk.size()];
		hipMemcpy(test_lk, dev_lk, host_lk.size()*sizeof(double), hipMemcpyDeviceToHost);
		for(size_t i=0;i<host_lk.size();i++){
			cout<<test_lk[i]<<'\t'<<host_lk[i]<<endl;
		}
		delete test_lk;
		*/

		cudaStatus = hipMemcpy(dev_pcp2M, &host_pcp2M[0], host_pcp2M.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMemcpy(dev_pcp2F, &host_pcp2F[0], host_pcp2F.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		/*
		double* test_pcp2M = new double[host_pcp2M.size()];
		hipMemcpy(test_pcp2M, dev_pcp2M, host_pcp2M.size()*sizeof(double),hipMemcpyDeviceToHost);
		for(size_t i=0;i<host_pcp2M.size();i++)
		{
			cout<<test_pcp2M[i]<<'\t'<<host_pcp2M[i]<<endl;
		}
		delete test_pcp2M;

		double* test_pcp2F = new double[host_pcp2F.size()];
		hipMemcpy(test_pcp2F, dev_pcp2F, host_pcp2F.size()*sizeof(double),hipMemcpyDeviceToHost);
		for(size_t i=0;i<host_pcp2F.size();i++)
		{
			cout<<test_pcp2F[i]<<'\t'<<host_pcp2F[i]<<endl;
		}
		delete test_pcp2F;
		*/
		

		cudaStatus = hipMemcpy(dev_genoFryF, &genoProbF[0], genoProbF.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}

		cudaStatus = hipMemcpy(dev_genoFryM, &genoProbM[0], genoProbM.size() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			flagPB=false;
			return false;
		}


/*
		// Allocate CUDA events that we'll use for timing
		hipError_t error;
		hipEvent_t start;
		error = hipEventCreate(&start);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		hipEvent_t stop;
		error = hipEventCreate(&stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(start, NULL);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		calPostProbX<<<NUM_BLOCK, NUM_THREAD>>>(dev_gender, dev_father, dev_mother, dev_lk, dev_pcp2M, dev_pcp2F, dev_postProb, dev_genoFryM, dev_genoFryF, numInd, numGeno);
		
				
		// Record the stop event
		error = hipEventRecord(stop, NULL);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Wait for the stop event to complete
		error = hipEventSynchronize(stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		float msecTotal = 0.0f;
		error = hipEventElapsedTime(&msecTotal, start, stop);

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Compute and print the performance
		printf("Time= %.3f msec\n",msecTotal);
*/		
		//sumPostProb<<<1,1>>>(dev_postProb, dev_posteriorProb, numGeno, numInd);
		
		calPostProbX<<<NUM_BLOCK, NUM_THREAD>>>(dev_gender, dev_father, dev_mother, dev_lk, dev_pcp2M, dev_pcp2F, dev_postProb, dev_genoFryM, dev_genoFryF, numInd, numGeno);

		double * host_postProb = new double[NUM_THREAD*NUM_BLOCK*3*numInd];
		cudaStatus = hipMemcpy(host_postProb, dev_postProb, NUM_THREAD*NUM_BLOCK*3*numInd*sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			hipFree((void*)dev_gender);
			hipFree((void*)dev_father);
			hipFree((void*)dev_mother);
			hipFree((void*)dev_lk);
			hipFree((void*)dev_pcp2M);
			hipFree((void*)dev_pcp2F);
			hipFree((void*)dev_postProb);
			hipFree((void*)dev_genoFryM);
			hipFree((void*)dev_genoFryF);

			delete host_postProb;

			flagPB=false;
			return false;
		}
		
		postProb.setZero();
		for(int i=0;i<NUM_BLOCK*NUM_THREAD;i++){
			for(int j=0;j<numInd;j++){
				for(int k=0;k<3;k++){
					postProb(j,k)=postProb(j,k)+host_postProb[i*numInd*3+j*3+k];
				}
			}
		}
		
		for(unsigned int i=0;i<numInd;i++){
			double sTmp = 0;
			for(int j=0;j<3;j++){
				sTmp = sTmp + postProb(i,j);
				//cout<<postProb(i,j)<<'\t';
			}
			//cout<<endl;
			if(sTmp<=0)
			{
				return false;
			}
			else
			{
				for(int j=0;j<3;j++){
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}

		hipFree((void*)dev_gender);
		hipFree((void*)dev_father);
		hipFree((void*)dev_mother);
		hipFree((void*)dev_lk);
		hipFree((void*)dev_pcp2M);
		hipFree((void*)dev_pcp2F);
		hipFree((void*)dev_postProb);
		hipFree((void*)dev_genoFryM);
		hipFree((void*)dev_genoFryF);

		delete host_postProb;
	}

	flagPB=true;
	return true;
}
#endif

#if 0
bool family::calPostProbBN(bool Known, int chrType)
{
	if(!flagLK)
	{
		cout<<"Likelihood has not been set. Please set likelihood first."<<endl;
		return false;
	}

	if(!calPostProbSingle(Known,chrType))
	{
		//cout<<Known<<'\t'<<chrType<<endl;
		//cout<<likelihood<<endl;
		return false;
	}

	//cout<<likelihood<<endl;

	bool flagSingle=true;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double big=0;
			double sum=0;
			for(int j=0;j<3;j++)
			{
				if(big<likelihood(mapV2P[i],j))
				{
					big=likelihood(mapV2P[i],j);
				}
				sum=sum+likelihood(mapV2P[i],j);
			}
			big=big/sum;
			if(big<m_lc)
			{
				flagSingle=false;
				break;
			}
		}
	}

	postProb.setZero();

	if(flagSingle)
	{
		if(chrType==0)
		{
			vector<double> genoProb;
			if(Known)
			{
				genoProb=genoProbK;
			}
			else
			{
				genoProb=genoProbN;
			}
			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					postProb(i,j)=likelihood(i,j)*genoProb[j];
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		else if(chrType==1)
		{
			vector<double> genoProbM;
			vector<double> genoProbF;
			if(Known)
			{
				genoProbM=genoProbXK;
			}
			else
			{
				genoProbM=genoProbXN;
			}
			if(Known)
			{
				genoProbF=genoProbK;
			}
			else
			{
				genoProbF=genoProbN;
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					if(member[i].get_gender()==1)
					{
						postProb(i,j)=likelihood(i,j)*genoProbM[j];
					}
					else
					{
						postProb(i,j)=likelihood(i,j)*genoProbF[j];
					}
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		flagPB=true;
		return true;
	}



	if(chrType==0)
	{
		vector<double> genoProb;
		if(Known)
		{
			genoProb=genoProbK;
		}
		else
		{
			genoProb=genoProbN;
		}

		vector<int> genotype(numInd,0);
		vector<double> indProb(numInd,0);

		while(true)
		{
			for(unsigned int i=0;i<numInd;i++)
			{
				if(parent[i].size()==0)
				{
					indProb[i]=genoProb[genotype[i]]*likelihood(i,genotype[i]);
				}
				else
				{
					indProb[i]=pcp2[genotype[i]](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,genotype[i]);
				}
			}

			double probAll=10000000;
			for(unsigned int i=0;i<numInd;i++)
			{
				//cout<<indProb[i]<<endl;
				probAll=probAll*indProb[i];
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				postProb(i,genotype[i])=postProb(i,genotype[i])+probAll;
			}

			unsigned int index=0;
			while(index<numInd)
			{
				genotype[index]=genotype[index]+1;
				if(genotype[index]==3)
				{
					genotype[index]=0;
					index++;
				}
				else
				{
					break;
				}
			}
			if(index==numInd)
			{
				break;
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			double sTmp=postProb.sum_row(i);
			if(sTmp<=0)
			{
				return false;
			}
			for(unsigned int j=0;j<3;j++)
			{
				postProb(i,j)=postProb(i,j)/sTmp;
			}
		}

		///////////////
//		cout<<postProb<<endl;
//		cout<<postProbSingle<<endl;
//		for(size_t ii=0;ii<child.size();ii++)
//		{
//			for(size_t jj=0;jj<child[ii].size();jj++)
//			{
//				cout<<child[ii][jj]<<'\t';
//			}
//			cout<<endl;
//		}
//		cout<<endl;
//
//		for(size_t ii=0;ii<parent.size();ii++)
//		{
//			for(size_t jj=0;jj<parent[ii].size();jj++)
//			{
//				cout<<parent[ii][jj]<<'\t';
//			}
//			cout<<endl;
//		}
//		cout<<endl;
//
//		for(size_t ii=0;ii<spouse.size();ii++)
//		{
//			for(size_t jj=0;jj<spouse[ii].size();jj++)
//			{
//				cout<<spouse[ii][jj]<<'\t';
//			}
//			cout<<endl;
//		}
//		cout<<endl;

	}
	else if(chrType==1)
	{
		vector<double> genoProbM;
		vector<double> genoProbF;
		if(Known)
		{
			genoProbM=genoProbXK;
		}
		else
		{
			genoProbM=genoProbXN;
		}
		if(Known)
		{
			genoProbF=genoProbK;
		}
		else
		{
			genoProbF=genoProbN;
		}

		vector<int> genotype(numInd,0);
		vector<double> indProb(numInd,0);

		while(true)
		{
			/*
			bool flagConti=false;
			for(unsigned int i=0;i<numInd;i++)
			{
				if(member[i].get_gender()==1 && genotype[i]==1)
				{
					flagConti=true;
					break;
				}
			}
			if(flagConti)
			{
				unsigned int index=0;
				while(index<numInd)
				{
					genotype[index]=genotype[index]+1;
					if(genotype[index]==3)
					{
						genotype[index]=0;
						index++;
					}
					else
					{
						break;
					}
				}
				if(index==numInd)
				{
					break;
				}
				continue;
			}
			*/

			for(unsigned int i=0;i<numInd;i++)
			{
				if(parent[i].size()==0)
				{
					if(member[i].get_gender()==1)
					{
						indProb[i]=genoProbM[genotype[i]]*likelihood(i,genotype[i]);
					}
					else
					{
						indProb[i]=genoProbF[genotype[i]]*likelihood(i,genotype[i]);
					}
				}
				else
				{
					if(member[i].get_gender()==1)
					{
						indProb[i]=pcp2Xm[genotype[i]](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,genotype[i]);
					}
					else
					{
						indProb[i]=pcp2Xf[genotype[i]](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,genotype[i]);
					}
				}
			}

			double probAll=10000000;
			for(unsigned int i=0;i<numInd;i++)
			{
				//cout<<indProb[i]<<endl;
				probAll=probAll*indProb[i];
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				postProb(i,genotype[i])=postProb(i,genotype[i])+probAll;
			}

			unsigned int index=0;
			while(index<numInd)
			{
				genotype[index]=genotype[index]+1;
				if(genotype[index]==3)
				{
					genotype[index]=0;
					index++;
				}
				else
				{
					break;
				}
			}
			if(index==numInd)
			{
				break;
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			double sTmp=postProb.sum_row(i);
			if(sTmp<=0)
			{
				return false;
			}
			for(unsigned int j=0;j<3;j++)
			{
				postProb(i,j)=postProb(i,j)/sTmp;
			}
		}
	}

	flagPB=true;
	return true;
}

#endif

bool family::calPostProbPeeling(bool Known, int chrType)
{
	if(!flagLK)
	{
		cout<<"Likelihood has not been set. Please set likelihood first."<<endl;
		return false;
	}

	if(!calPostProbSingle(Known,chrType))
	{
		return false;
	}

	//calPostProbSingle();
	bool flagSingle=true;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double big=0;
			double sum=0;
			for(int j=0;j<3;j++)
			{
				if(big<likelihood(mapV2P[i],j))
				{
					big=likelihood(mapV2P[i],j);
				}
				sum=sum+likelihood(mapV2P[i],j);
			}
			big=big/sum;
			if(big<m_lc)
			{
				flagSingle=false;
				break;
			}
		}
	}

	postProb.setZero();

	//flagSingle=false;

	if(flagSingle)
	{
		if(chrType==0)
		{
			vector<double> genoProb;
			if(Known)
			{
				genoProb=genoProbK;
			}
			else
			{
				genoProb=genoProbN;
			}
			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					postProb(i,j)=likelihood(i,j)*genoProb[j];
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		else if(chrType==1)
		{
			vector<double> genoProbM;
			vector<double> genoProbF;
			if(Known)
			{
				genoProbM=genoProbXK;
			}
			else
			{
				genoProbM=genoProbXN;
			}
			if(Known)
			{
				genoProbF=genoProbK;
			}
			else
			{
				genoProbF=genoProbN;
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					if(member[i].get_gender()==1)
					{
						postProb(i,j)=likelihood(i,j)*genoProbM[j];
					}
					else
					{
						postProb(i,j)=likelihood(i,j)*genoProbF[j];
					}
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		flagPB=true;
		return true;
	}

	if(chrType==0)
	{
		vector<double> genoProb;
		if(Known)
		{
			genoProb=genoProbK;
		}
		else
		{
			genoProb=genoProbN;
		}

		dMatrix<double> antProb(numInd,3,-1);
		//posProb[iInd](jInd,iGeno)
		//iInd: index of i
		//jInd: index of j (i's spouse)
		//iGeno: i's genotype
		vector<dMatrix<double> > posProb;
		for(unsigned int i=0;i<numInd;i++)
		{
			dMatrix<double> dMTmp(numInd,3,-1);
			posProb.push_back(dMTmp);
		}

		//prepare anterior probability
		//set founder's anterior probability
		for(unsigned int i=0;i<numInd;i++)
		{
			if(parent[i].size()==0)
			{
				for(unsigned int j=0;j<3;j++)
				{
					antProb(i,j)=genoProb[j];
				}
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			for(unsigned int j=0;j<3;j++)
			{
				double postTmp=1;
				for(unsigned int k=0;k<spouse[i].size();k++)
				{
					postTmp=postTmp*calPosProb(i,j,spouse[i][k],antProb,posProb);
				}
				postTmp=postTmp*likelihood(i,j)*calAntProb(i,j,antProb,posProb);
				postProb(i,j)=postTmp;
			}

			double sTmp=postProb.sum_row(i);
			if(sTmp==0)
			{
				//cout<<antProb<<endl;
				return false;
			}
			for(unsigned int j=0;j<3;j++)
			{
				postProb(i,j)=postProb(i,j)/sTmp;
			}
		}
	}
	else if(chrType==1)
	{
		vector<double> genoProbM;
		vector<double> genoProbF;
		if(Known)
		{
			genoProbM=genoProbXK;
		}
		else
		{
			genoProbM=genoProbXN;
		}
		if(Known)
		{
			genoProbF=genoProbK;
		}
		else
		{
			genoProbF=genoProbN;
		}

		dMatrix<double> antProb(numInd,3,-1);
		//posProb[iInd](jInd,iGeno)
		//iInd: index of i
		//jInd: index of j (i's spouse)
		//iGeno: i's genotype
		vector<dMatrix<double> > posProb;
		for(unsigned int i=0;i<numInd;i++)
		{
			dMatrix<double> dMTmp(numInd,3,-1);
			posProb.push_back(dMTmp);
		}

		//prepare anterior probability
		//set founder's anterior probability
		for(unsigned int i=0;i<numInd;i++)
		{
			if(parent[i].size()==0)
			{
				if(member[i].get_gender()==1)
				{
					for(unsigned int j=0;j<3;j++)
					{
						antProb(i,j)=genoProbM[j];
					}
				}
				else
				{
					for(unsigned int j=0;j<3;j++)
					{
						antProb(i,j)=genoProbF[j];
					}
				}
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			for(int j=0;j<3;j++)
			{
				double postTmp=1;
				for(unsigned int k=0;k<spouse[i].size();k++)
				{
					postTmp=postTmp*calPosProbX(i,j,spouse[i][k],antProb,posProb);
				}
				postTmp=postTmp*likelihood(i,j)*calAntProbX(i,j,antProb,posProb);
				postProb(i,j)=postTmp;
			}

			double sTmp=postProb.sum_row(i);
			if(sTmp==0)
			{
				//cout<<antProb<<endl;
				return false;
			}
			for(int j=0;j<3;j++)
			{
				postProb(i,j)=postProb(i,j)/sTmp;
			}
		}
	}
	else
	{
	}
	flagPB=true;
	return true;
}

bool family::calPostProbSingle(bool Known, int chrType)
{
	if(!flagLK)
	{
		cout<<"Likelihood has not been set. Please set likelihood first."<<endl;
		return false;
	}

	postProbSingle.setZero();

	if(chrType==0)
	{
		vector<double> genoProb;
		if(Known)
		{
			genoProb=genoProbK;
		}
		else
		{
			genoProb=genoProbN;
		}
		for(unsigned int i=0;i<numInd;i++)
		{
			for(int j=0;j<3;j++)
			{
				postProbSingle(i,j)=likelihood(i,j)*genoProb[j];
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			double sTmp=postProbSingle.sum_row(i);
			if(sTmp<=0)
			{
				return false;
			}
			for(unsigned int j=0;j<3;j++)
			{
				postProbSingle(i,j)=postProbSingle(i,j)/sTmp;
			}
		}
	}
	else if(chrType==1)
	{
		vector<double> genoProbM;
		vector<double> genoProbF;
		if(Known)
		{
			genoProbM=genoProbXK;
		}
		else
		{
			genoProbM=genoProbXN;
		}
		if(Known)
		{
			genoProbF=genoProbK;
		}
		else
		{
			genoProbF=genoProbN;
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			for(int j=0;j<3;j++)
			{
				if(member[i].get_gender()==1)
				{
					postProbSingle(i,j)=likelihood(i,j)*genoProbM[j];
				}
				else
				{
					postProbSingle(i,j)=likelihood(i,j)*genoProbF[j];
				}
			}
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			double sTmp=postProbSingle.sum_row(i);
			if(sTmp<=0)
			{
				return false;
			}
			for(unsigned int j=0;j<3;j++)
			{
				postProbSingle(i,j)=postProbSingle(i,j)/sTmp;
			}
		}
	}

	flagPBS=true;
	return true;
}

double family::calAntProb(int iInd, int iGeno, dMatrix<double> & antProb, vector<dMatrix<double> > & posProb)
{
	//double returnVal=0;

	if(antProb(iInd,iGeno)>=0)
	{
		return antProb(iInd,iGeno);
	}

	//unsigned int numGeno=priorProb.get_column();

	/*
	if(parent[iInd].size()==1)
	{
		int p1=parent[iInd][0];

		//other child beside iInd
		vector<int> otherChild;
		for(unsigned int i=0;i<child[p1].size();i++)
		{
			if(child[p1][i]!=iInd && parent[child[p1][i]].size()==1)
			{
				otherChild.push_back(child[p1][i]);
			}
		}

		//other spouse of p1
		vector<int> otherSpouse1=spouse[p1];

		double sm=0;
		for(unsigned int i=0;i<3;i++)
		{
			double sf=0;
			for(unsigned int j=0;j<numGeno;j++)
			{
				double mc=1;
				for(unsigned int k=0;k<otherChild.size();k++)
				{
					double sc=0;
					for(unsigned int l=0;l<numGeno;l++)
					{
						double mcs=1;
						for(unsigned int m=0;m<spouse[otherChild[k]].size();m++)
						{
							mcs=mcs*calPosProb(otherChild[k],l,spouse[otherChild[k]][m],antProb,posProb);
						}
						sc=sc+mcs*likelihood(otherChild[k],l)*pcp2[l](i,j);
					}
					mc=mc*sc;
				}

				//no other spouse for p2
				double mf=1;


				sf=sf+genoProb[j]*mf*pcp2[iGeno](i,j)*mc/numGeno;
			}
			double mm=1;
			for(unsigned int j=0;j<otherSpouse1.size();j++)
			{
				mm=mm*calPosProb(p1,i,otherSpouse1[j],antProb,posProb);
			}
			sm=sm+calAntProb(p1,i,antProb,posProb)*likelihood(p1,i)*mm*sf;
		}

		antProb(iInd,iGeno)=sm;
		return sm;
	}
	else
	*/
	{
		int p1=parent[iInd][0];
		int p2=parent[iInd][1];

		//other child beside iInd
		vector<int> otherChild;
		for(unsigned int i=0;i<child[p1].size();i++)
		{
			for(unsigned int j=0;j<child[p2].size();j++)
			{
				if(child[p1][i]==child[p2][j] && child[p1][i]!=iInd)
				{
					otherChild.push_back(child[p1][i]);
					break;
				}
			}
		}

		//p1's other spouse beside p2
		vector<int> otherSpouse1;
		for(unsigned int i=0;i<spouse[p1].size();i++)
		{
			if(spouse[p1][i]!=p2)
			{
				otherSpouse1.push_back(spouse[p1][i]);
			}
		}

		//p2's other spouse beside p1
		vector<int> otherSpouse2;
		for(unsigned int i=0;i<spouse[p2].size();i++)
		{
			if(spouse[p2][i]!=p1)
			{
				otherSpouse2.push_back(spouse[p2][i]);
			}
		}

		double sm=0;
		for(unsigned int i=0;i<3;i++)
		{
			double sf=0;
			for(unsigned int j=0;j<3;j++)
			{
				double mc=1;
				for(unsigned int k=0;k<otherChild.size();k++)
				{
					double sc=0;
					for(unsigned int l=0;l<3;l++)
					{
						double mcs=1;
						for(unsigned int m=0;m<spouse[otherChild[k]].size();m++)
						{
							mcs=mcs*calPosProb(otherChild[k],l,spouse[otherChild[k]][m],antProb,posProb);
						}
						sc=sc+mcs*likelihood(otherChild[k],l)*pcp2[l](i,j);
					}
					mc=mc*sc;
				}

				double mf=1;
				for(unsigned int k=0;k<otherSpouse2.size();k++)
				{
					mf=mf*calPosProb(p2,j,otherSpouse2[k],antProb,posProb);
				}
				sf=sf+calAntProb(p2,j,antProb,posProb)*likelihood(p2,j)*mf*pcp2[iGeno](i,j)*mc;
			}
			double mm=1;
			for(unsigned int j=0;j<otherSpouse1.size();j++)
			{
				mm=mm*calPosProb(p1,i,otherSpouse1[j],antProb,posProb);
			}
			sm=sm+calAntProb(p1,i,antProb,posProb)*likelihood(p1,i)*mm*sf;
		}

		antProb(iInd,iGeno)=sm;
		return sm;
	}
}

double family::calAntProbX(int iInd, int iGeno, dMatrix<double> & antProb, vector<dMatrix<double> > & posProb)
{
	//double returnVal=0;

	if(antProb(iInd,iGeno)>=0)
	{
		return antProb(iInd,iGeno);
	}

	//unsigned int numGeno=priorProb.get_column();
	{
		int p1=parent[iInd][0];
		int p2=parent[iInd][1];

		//other child beside iInd
		vector<int> otherChild;
		for(unsigned int i=0;i<child[p1].size();i++)
		{
			for(unsigned int j=0;j<child[p2].size();j++)
			{
				if(child[p1][i]==child[p2][j] && child[p1][i]!=iInd)
				{
					otherChild.push_back(child[p1][i]);
					break;
				}
			}
		}

		//p1's other spouse beside p2
		vector<int> otherSpouse1;
		for(unsigned int i=0;i<spouse[p1].size();i++)
		{
			if(spouse[p1][i]!=p2)
			{
				otherSpouse1.push_back(spouse[p1][i]);
			}
		}

		//p2's other spouse beside p1
		vector<int> otherSpouse2;
		for(unsigned int i=0;i<spouse[p2].size();i++)
		{
			if(spouse[p2][i]!=p1)
			{
				otherSpouse2.push_back(spouse[p2][i]);
			}
		}

		double sm=0;
		for(unsigned int i=0;i<3;i++)
		{
			double sf=0;
			for(unsigned int j=0;j<3;j++)
			{
				double mc=1;
				for(unsigned int k=0;k<otherChild.size();k++)
				{
					double sc=0;
					for(unsigned int l=0;l<3;l++)
					{
						double mcs=1;
						for(unsigned int m=0;m<spouse[otherChild[k]].size();m++)
						{
							mcs=mcs*calPosProbX(otherChild[k],l,spouse[otherChild[k]][m],antProb,posProb);
						}
						if(member[otherChild[k]].get_gender()==1)
						{
							if(member[p1].get_gender()==1)
							{
								sc=sc+mcs*likelihood(otherChild[k],l)*pcp2Xm[l](j,i);
							}
							else
							{
								sc=sc+mcs*likelihood(otherChild[k],l)*pcp2Xm[l](i,j);
							}
						}
						else
						{
							if(member[p1].get_gender()==1)
							{
								sc=sc+mcs*likelihood(otherChild[k],l)*pcp2Xf[l](j,i);
							}
							else
							{
								sc=sc+mcs*likelihood(otherChild[k],l)*pcp2Xf[l](i,j);
							}
						}
					}
					mc=mc*sc;
				}

				double mf=1;
				for(unsigned int k=0;k<otherSpouse2.size();k++)
				{
					mf=mf*calPosProbX(p2,j,otherSpouse2[k],antProb,posProb);
				}
				if(member[iInd].get_gender()==1)
				{
					if(member[p1].get_gender()==1)
					{
						sf=sf+calAntProbX(p2,j,antProb,posProb)*likelihood(p2,j)*mf*pcp2Xm[iGeno](j,i)*mc;
					}
					else
					{
						sf=sf+calAntProbX(p2,j,antProb,posProb)*likelihood(p2,j)*mf*pcp2Xm[iGeno](i,j)*mc;
					}
				}
				else
				{
					if(member[p1].get_gender()==1)
					{
						sf=sf+calAntProbX(p2,j,antProb,posProb)*likelihood(p2,j)*mf*pcp2Xf[iGeno](j,i)*mc;
					}
					else
					{
						sf=sf+calAntProbX(p2,j,antProb,posProb)*likelihood(p2,j)*mf*pcp2Xf[iGeno](i,j)*mc;
					}
				}
			}
			double mm=1;
			for(unsigned int j=0;j<otherSpouse1.size();j++)
			{
				mm=mm*calPosProbX(p1,i,otherSpouse1[j],antProb,posProb);
			}
			sm=sm+calAntProbX(p1,i,antProb,posProb)*likelihood(p1,i)*mm*sf;
		}

		antProb(iInd,iGeno)=sm;
		return sm;
	}
}

double family::calPosProb(int iInd, int iGeno, int jInd, dMatrix<double> & antProb, vector<dMatrix<double> > & posProb)
{
	//double returnVal=0;

	if(posProb[iInd](jInd,iGeno)>=0)
	{
		return posProb[iInd](jInd,iGeno);
	}

	//unsigned int numGeno=priorProb.get_column();

	//jInd's other spouse beside iInd
	vector<int> otherSpouse;
	for(unsigned int i=0;i<spouse[jInd].size();i++)
	{
		if(spouse[jInd][i]!=iInd)
		{
			otherSpouse.push_back(spouse[jInd][i]);
		}
	}

	//iInd and jInd's child
	vector<int> allChild;
	for(unsigned int i=0;i<child[iInd].size();i++)
	{
		for(unsigned int j=0;j<child[jInd].size();j++)
		{
			if(child[iInd][i]==child[jInd][j])
			{
				allChild.push_back(child[iInd][i]);
			}
		}
	}

	double sj=0;
	for(unsigned int i=0;i<3;i++)
	{
		double ms=1;
		for(unsigned int j=0;j<otherSpouse.size();j++)
		{
			ms=ms*calPosProb(jInd,i,otherSpouse[j],antProb,posProb);
		}
		double mc=1;
		for(unsigned int j=0;j<allChild.size();j++)
		{
			double sc=0;
			for(unsigned int k=0;k<3;k++)
			{
				double mcs=1;
				for(unsigned int l=0;l<spouse[allChild[j]].size();l++)
				{
					mcs=mcs*calPosProb(allChild[j],k,spouse[allChild[j]][l],antProb,posProb);
				}
				sc=sc+pcp2[k](iGeno,i)*likelihood(allChild[j],k)*mcs;
			}
			mc=mc*sc;
		}
		sj=sj+calAntProb(jInd,i,antProb,posProb)*likelihood(jInd,i)*ms*mc;
	}

	posProb[iInd](jInd,iGeno)=sj;
	return sj;
}

double family::calPosProbX(int iInd, int iGeno, int jInd, dMatrix<double> & antProb, vector<dMatrix<double> > & posProb)
{
	//double returnVal=0;

	if(posProb[iInd](jInd,iGeno)>=0)
	{
		return posProb[iInd](jInd,iGeno);
	}

	//unsigned int numGeno=priorProb.get_column();

	//jInd's other spouse beside iInd
	vector<int> otherSpouse;
	for(unsigned int i=0;i<spouse[jInd].size();i++)
	{
		if(spouse[jInd][i]!=iInd)
		{
			otherSpouse.push_back(spouse[jInd][i]);
		}
	}

	//iInd and jInd's child
	vector<int> allChild;
	for(unsigned int i=0;i<child[iInd].size();i++)
	{
		for(unsigned int j=0;j<child[jInd].size();j++)
		{
			if(child[iInd][i]==child[jInd][j])
			{
				allChild.push_back(child[iInd][i]);
			}
		}
	}

	double sj=0;
	for(unsigned int i=0;i<3;i++)
	{
		double ms=1;
		for(unsigned int j=0;j<otherSpouse.size();j++)
		{
			ms=ms*calPosProbX(jInd,i,otherSpouse[j],antProb,posProb);
		}
		double mc=1;
		for(unsigned int j=0;j<allChild.size();j++)
		{
			double sc=0;
			for(unsigned int k=0;k<3;k++)
			{
				double mcs=1;
				for(unsigned int l=0;l<spouse[allChild[j]].size();l++)
				{
					mcs=mcs*calPosProbX(allChild[j],k,spouse[allChild[j]][l],antProb,posProb);
				}
				if(member[allChild[j]].get_gender()==1)
				{
					if(member[iInd].get_gender()==1)
					{
						sc=sc+pcp2Xm[k](i,iGeno)*likelihood(allChild[j],k)*mcs;
					}
					else
					{
						sc=sc+pcp2Xm[k](iGeno,i)*likelihood(allChild[j],k)*mcs;
					}
				}
				else
				{
					if(member[iInd].get_gender()==1)
					{
						sc=sc+pcp2Xf[k](i,iGeno)*likelihood(allChild[j],k)*mcs;
					}
					else
					{
						sc=sc+pcp2Xf[k](iGeno,i)*likelihood(allChild[j],k)*mcs;
					}
				}
			}
			mc=mc*sc;
		}
		sj=sj+calAntProbX(jInd,i,antProb,posProb)*likelihood(jInd,i)*ms*mc;
	}

	posProb[iInd](jInd,iGeno)=sj;
	return sj;
}

bool family::calPostProbMCMC(int numBurnIn, int numRep,bool Known,int chrType)
{
	if(!flagLK)
	{
		cout<<"Likelihood has not been set. Please set likelihood first."<<endl;
		return false;
	}

	if(!calPostProbSingle(Known,chrType))
	{
		//cout<<Known<<'\t'<<chrType<<endl;
		//cout<<likelihood<<endl;
		return false;
	}

	//cout<<likelihood<<endl;

	bool flagSingle=true;
	for(size_t i=0;i<mapV2P.size();i++)
	{
		if(mapV2P[i]>=0)
		{
			double big=0;
			double sum=0;
			for(int j=0;j<3;j++)
			{
				if(big<likelihood(mapV2P[i],j))
				{
					big=likelihood(mapV2P[i],j);
				}
				sum=sum+likelihood(mapV2P[i],j);
			}
			big=big/sum;
			if(big<m_lc)
			{
				flagSingle=false;
				break;
			}
		}
	}

	postProb.setZero();

	if(flagSingle)
	{
		if(chrType==0)
		{
			vector<double> genoProb;
			if(Known)
			{
				genoProb=genoProbK;
			}
			else
			{
				genoProb=genoProbN;
			}
			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					postProb(i,j)=likelihood(i,j)*genoProb[j];
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		else if(chrType==1)
		{
			vector<double> genoProbM;
			vector<double> genoProbF;
			if(Known)
			{
				genoProbM=genoProbXK;
			}
			else
			{
				genoProbM=genoProbXN;
			}
			if(Known)
			{
				genoProbF=genoProbK;
			}
			else
			{
				genoProbF=genoProbN;
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				for(int j=0;j<3;j++)
				{
					if(member[i].get_gender()==1)
					{
						postProb(i,j)=likelihood(i,j)*genoProbM[j];
					}
					else
					{
						postProb(i,j)=likelihood(i,j)*genoProbF[j];
					}
				}
			}

			for(unsigned int i=0;i<numInd;i++)
			{
				double sTmp=postProb.sum_row(i);
				if(sTmp<=0)
				{
					return false;
				}
				for(unsigned int j=0;j<3;j++)
				{
					postProb(i,j)=postProb(i,j)/sTmp;
				}
			}
		}
		flagPB=true;
		return true;
	}


	vector<int> genotype(numInd,0);
	for(unsigned int i=0;i<numInd;i++)
	{
		genotype[i]=rand()%3;
	}

	dMatrix<double> genoFry(numInd,3,0);
	for(int i=0;i<numBurnIn;i++)
	{
		estGenoProb(genotype,genoFry,Known,chrType);
	}

	genoFry.setZero();

	for(int i=0;i<numRep;i++)
	{
		estGenoProb(genotype,genoFry,Known,chrType);
	}

	for(unsigned int i=0;i<numInd;i++)
	{
		for(int j=0;j<3;j++)
		{
			postProb(i,j)=genoFry(i,j)/numRep;
		}
		if(postProb.sum_row(i)<=0)
		{
			return false;
		}
	}

	flagPB=true;
	return true;
}

bool family::estGenoProb(std::vector<int> & genotype, dMatrix<double> & genoFre,bool Known,int chrType)
{
	//normal chromosome
	if(chrType==0)
	{
		vector<double> genoProb;
		if(Known)
		{
			genoProb=genoProbK;
		}
		else
		{
			genoProb=genoProbN;
		}
		//cout<<likelihood<<endl;
		for(unsigned int i=0;i<numInd;i++)
		{
			vector<double> genoFreTmp(3,1000000);
			for(int j=0;j<3;j++)
			{
				//cout<<genoFreTmp[j]<<endl;
				if(parent[i].size()==0)
				{
					genoFreTmp[j]=genoFreTmp[j]*genoProb[j]*likelihood(i,j);
				}
				else
				{
					genoFreTmp[j]=genoFreTmp[j]*pcp2[j](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,j);
				}
				//cout<<genoFreTmp[j]<<endl;
				for(size_t k=0;k<child[i].size();k++)
				{
					int indChild=child[i][k];
					if(member[i].get_gender()==1)
					{
						genoFreTmp[j]=genoFreTmp[j]*pcp2[genotype[indChild]](genotype[parent[indChild][0]],j);
					}
					else
					{
						genoFreTmp[j]=genoFreTmp[j]*pcp2[genotype[indChild]](j,genotype[parent[indChild][1]]);
					}
					//cout<<genoFreTmp[j]<<endl;
				}
			}
			double sFre=0;
			for(int j=0;j<3;j++)
			{
				sFre=sFre+genoFreTmp[j];
			}
			if(sFre<=0)
			{
				for(int j=0;j<3;j++)
				{
					genoFreTmp[j]=0;
				}
			}
			else
			{
				for(int j=0;j<3;j++)
				{
					genoFreTmp[j]=genoFreTmp[j]/sFre;
				}
			}
			double rd=double(rand())/double(RAND_MAX);
			if(rd<genoFreTmp[0])
			{
				genotype[i]=0;
			}
			else if(rd>(1.0-genoFreTmp[2]))
			{
				genotype[i]=2;
			}
			else
			{
				genotype[i]=1;
			}

			for(int j=0;j<3;j++)
			{
				genoFre(i,j)=genoFre(i,j)+genoFreTmp[j];
			}
		}
	}

	//chromosome X
	else if(chrType==1)
	{
		vector<double> genoProbM;
		vector<double> genoProbF;
		if(Known)
		{
			genoProbM=genoProbXK;
			genoProbF=genoProbK;
		}
		else
		{
			genoProbM=genoProbXN;
			genoProbF=genoProbN;
		}

		for(unsigned int i=0;i<numInd;i++)
		{
			vector<double> genoFreTmp(3,1000000);
			if(member[i].get_gender()==1)
			{
				for(int j=0;j<3;j++)
				{
					if(parent[i].size()==0)
					{
						genoFreTmp[j]=genoFreTmp[j]*genoProbM[j]*likelihood(i,j);
					}
					else
					{
						genoFreTmp[j]=genoFreTmp[j]*pcp2Xm[j](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,j);
					}
					for(size_t k=0;k<child[i].size();k++)
					{
						int indChild=child[i][k];
						if(member[i].get_gender()==1)
						{
							if(member[indChild].get_gender()==1)
							{
								genoFreTmp[j]=genoFreTmp[j]*pcp2Xm[genotype[indChild]](genotype[parent[indChild][0]],j);
							}
							else
							{
								genoFreTmp[j]=genoFreTmp[j]*pcp2Xf[genotype[indChild]](genotype[parent[indChild][0]],j);
							}
						}
					}
				}
			}
			else
			{
				for(int j=0;j<3;j++)
				{
					if(parent[i].size()==0)
					{
						genoFreTmp[j]=genoFreTmp[j]*genoProbF[j]*likelihood(i,j);
					}
					else
					{
						genoFreTmp[j]=genoFreTmp[j]*pcp2Xf[j](genotype[parent[i][0]],genotype[parent[i][1]])*likelihood(i,j);
					}
					for(size_t k=0;k<child[i].size();k++)
					{
						int indChild=child[i][k];
						if(member[i].get_gender()==1)
						{
							if(member[indChild].get_gender()==1)
							{
								genoFreTmp[j]=genoFreTmp[j]*pcp2Xm[indChild](j,genotype[parent[indChild][1]]);
							}
							else
							{
								genoFreTmp[j]=genoFreTmp[j]*pcp2Xf[indChild](j,genotype[parent[indChild][1]]);
							}
						}
					}
				}
			}
			double sFre=0;
			for(int j=0;j<3;j++)
			{
				sFre=sFre+genoFreTmp[j];
			}
			if(sFre<=0)
			{
				for(int j=0;j<3;j++)
				{
					genoFreTmp[j]=0;
				}
			}
			else
			{
				for(int j=0;j<3;j++)
				{
					genoFreTmp[j]=genoFreTmp[j]/sFre;
				}
			}
			double rd=double(rand())/double(RAND_MAX);
			if(rd<genoFreTmp[0])
			{
				genotype[i]=0;
			}
			else if(rd>(1.0-genoFreTmp[2]))
			{
				genotype[i]=2;
			}
			else
			{
				genotype[i]=1;
			}

			for(int j=0;j<3;j++)
			{
				genoFre(i,j)=genoFre(i,j)+genoFreTmp[j];
			}
		}
	}
	return true;
}
